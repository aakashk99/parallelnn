#include "hip/hip_runtime.h"
#include <stdio.h>

extern "C" {
#include "ops.h"
Matrix* matrix_create(int row, int col);
}

__global__ void MatScale(double* a, double* c, int m, int n, double factor) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < m && col < n)
		c[row*n + col] = a[row*n + col] * factor; 
}

extern "C" Matrix* scale(double factor, Matrix* mat) {
	int m = mat->rows, n = mat->cols;

	double *d_a, *d_c;
	hipMalloc((void **) &d_a, sizeof(double)*m*n);
	hipMalloc((void **) &d_c, sizeof(double)*m*n);
	
	hipMemcpy(d_a, mat->entries, sizeof(double)*m*n, hipMemcpyHostToDevice);

	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	MatScale<<<dimGrid, dimBlock>>>(d_a, d_c, m, n, factor);

	Matrix* res = matrix_create(m, n);
	hipMemcpy(res->entries, d_c, sizeof(double)*m*n, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_c);

	return res;
}
